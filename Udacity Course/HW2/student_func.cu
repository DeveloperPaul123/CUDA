#include "hip/hip_runtime.h"
// Homework 2
// Image Blurring
//
// In this homework we are blurring an image. To do this, imagine that we have
// a square array of weight values. For each pixel in the image, imagine that we
// overlay this square array of weights on top of the image such that the center
// of the weight array is aligned with the current pixel. To compute a blurred
// pixel value, we multiply each pair of numbers that line up. In other words, we
// multiply each weight with the pixel underneath it. Finally, we add up all of the
// multiplied numbers and assign that value to our output for the current pixel.
// We repeat this process for all the pixels in the image.

// To help get you started, we have included some useful notes here.

//****************************************************************************

// For a color image that has multiple channels, we suggest separating
// the different color channels so that each color is stored contiguously
// instead of being interleaved. This will simplify your code.

// That is instead of RGBARGBARGBARGBA... we suggest transforming to three
// arrays (as in the previous homework we ignore the alpha channel again):
//  1) RRRRRRRR...
//  2) GGGGGGGG...
//  3) BBBBBBBB...
//
// The original layout is known an Array of Structures (AoS) whereas the
// format we are converting to is known as a Structure of Arrays (SoA).

// As a warm-up, we will ask you to write the kernel that performs this
// separation. You should then write the "meat" of the assignment,
// which is the kernel that performs the actual blur. We provide code that
// re-combines your blurred results for each color channel.

//****************************************************************************

// You must fill in the gaussian_blur kernel to perform the blurring of the
// inputChannel, using the array of weights, and put the result in the outputChannel.

// Here is an example of computing a blur, using a weighted average, for a single
// pixel in a small image.
//
// Array of weights:
//
//  0.0  0.2  0.0
//  0.2  0.2  0.2
//  0.0  0.2  0.0
//
// Image (note that we align the array of weights to the center of the box):
//
//    1  2  5  2  0  3
//       -------
//    3 |2  5  1| 6  0       0.0*2 + 0.2*5 + 0.0*1 +
//      |       |
//    4 |3  6  2| 1  4   ->  0.2*3 + 0.2*6 + 0.2*2 +   ->  3.2
//      |       |
//    0 |4  0  3| 4  2       0.0*4 + 0.2*0 + 0.0*3
//       -------
//    9  6  5  0  3  9
//
//         (1)                         (2)                 (3)
//
// A good starting place is to map each thread to a pixel as you have before.
// Then every thread can perform steps 2 and 3 in the diagram above
// completely independently of one another.

// Note that the array of weights is square, so its height is the same as its width.
// We refer to the array of weights as a filter, and we refer to its width with the
// variable filterWidth.

//****************************************************************************

// Your homework submission will be evaluated based on correctness and speed.
// We test each pixel against a reference solution. If any pixel differs by
// more than some small threshold value, the system will tell you that your
// solution is incorrect, and it will let you try again.

// Once you have gotten that working correctly, then you can think about using
// shared memory and having the threads cooperate to achieve better performance.

//****************************************************************************

// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//
// Writing code the safe way requires slightly more typing, but is very helpful for
// catching mistakes. If you write code the unsafe way and you make a mistake, then
// any subsequent kernels won't compute anything, and it will be hard to figure out
// why. Writing code the safe way will inform you as soon as you make a mistake.

// Finally, remember to free the memory you allocate at the end of the function.

//****************************************************************************

#include "utils.h"

#ifndef __HIPCC__
#define __HIPCC__
#endif

__global__ void gaussian_blur(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols,
                   const float* const filter, const int filterWidth)
{

	//so filter width defines width of the filter.
	assert(filterWidth % 2 == 1); //filter size should be odd.
	int r = blockIdx.x;
	int c = threadIdx.x;
	float result = 0.0f;
	if (r >= numRows || c >= numCols) {
		return;
	}
	int pos = r*numCols + c;
	for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; ++filter_r) {
		for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; ++filter_c) {
			//Find the global image position for this filter position
			//clamp to boundary of the image
			int rowCompare = r + filter_r >= 0 ? r + filter_r : 0;
			int colCompare = c + filter_c >= 0 ? c + filter_c : 0;

			int image_r = rowCompare <= static_cast<int>(numRows - 1) ? rowCompare : static_cast<int>(numRows - 1);
			int image_c = colCompare <= static_cast<int>(numCols - 1) ? colCompare : static_cast<int>(numCols - 1);

			float image_value = static_cast<float>(inputChannel[image_r * numCols + image_c]);
			float filter_value = filter[(filter_r + filterWidth / 2) * filterWidth + filter_c + filterWidth / 2];

			result += image_value*filter_value;
		}
	}
	outputChannel[pos] = result;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__ void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
	
	int row = blockIdx.x;
	int col = threadIdx.x;
	//check index, make sure it's not out of bounds.
	if (row >= numRows || col >= numCols) return;	
	int index = row*numCols + col;

	uchar4 val = inputImageRGBA[index];
	
	redChannel[index] = val.x;
	greenChannel[index] = val.y;
	blueChannel[index] = val.z;
}

//This kernel takes in three color channels and recombines them
//into one image.  The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__ void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
	if (blockIdx.x >= numRows || threadIdx.x >= numCols) return;
	int index = blockIdx.x*numCols + threadIdx.x;
	unsigned char red = redChannel[index];
	unsigned char green = greenChannel[index];
	unsigned char blue = blueChannel[index];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[index] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

/**
* Allocated memory and copies pointers/data arrays to the GPU.
* @param numRowsImage the number of rows in the image. 
* @param numColsImage the number of cols in the image.
* @param h_filter host pointer to the filter array
* @param filterWidth the width (size) of the filter array. Assumed to be square. 
*/
void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  //original
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //Allocate memory for the filter on the GPU
  //Use the pointer d_filter that we have already declared for you
  //You need to allocate memory for the filter with hipMalloc
  //be sure to use checkCudaErrors like the above examples to
  //be able to tell if anything goes wrong
  //IMPORTANT: Notice that we pass a pointer to a pointer to hipMalloc
  size_t size = sizeof(float) * filterWidth * filterWidth;
  checkCudaErrors(hipMalloc(&d_filter, size));

  //Copy the filter on the host (h_filter) to the memory you just allocated
  //on the GPU.  hipMemcpy(dst, src, numBytes, hipMemcpyHostToDevice);
  //Remember to use checkCudaErrors!
  checkCudaErrors(hipMemcpy(d_filter, h_filter, size, hipMemcpyHostToDevice));

}

void your_gaussian_blur(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redBlurred, 
                        unsigned char *d_greenBlurred, 
                        unsigned char *d_blueBlurred,
                        const int filterWidth)
{
  //TODO: Set reasonable block size (i.e., number of threads per block)
	const dim3 blockSize(numCols);

  //TODO:
  //Compute correct grid size (i.e., number of blocks per kernel launch)
  //from the image size and and block size.
	const dim3 gridSize(numRows);
  //Launch a kernel for separating the RGBA image into different color channels
  //block size is rows. Grid size is columns.
	separateChannels << < numRows, numCols>> >(d_inputImageRGBA, numRows, numCols, d_red, d_green, d_blue);

  std::cout << "Separated channels." << std::endl;
  // Call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  gaussian_blur << <gridSize, blockSize>> >(d_red, d_redBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur << <gridSize, blockSize >> > (d_green, d_greenBlurred, numRows, numCols, d_filter, filterWidth);
  gaussian_blur << <gridSize, blockSize >> > (d_blue, d_blueBlurred, numRows, numCols, d_filter, filterWidth);

  // Again, call hipDeviceSynchronize(), then call checkCudaErrors() immediately after
  // launching your kernel to make sure that you didn't make any mistakes.
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

  // Now we recombine your results. We take care of launching this kernel for you.
  //
  // NOTE: This kernel launch depends on the gridSize and blockSize variables,
  // which you must set yourself.
  recombineChannels<<<gridSize, blockSize>>>(d_redBlurred,
                                             d_greenBlurred,
                                             d_blueBlurred,
                                             d_outputImageRGBA,
                                             numRows,
                                             numCols);
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}
